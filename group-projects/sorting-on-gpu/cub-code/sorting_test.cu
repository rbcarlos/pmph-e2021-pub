#include "hip/hip_runtime.h"
//#include "../../cub-1.8.0/hipcub/hipcub.hpp"   // or equivalently <cub/device/device_histogram.cuh>
#include "cub.cuh"
#include "helper.cu.h"

template<class Z>
bool validateZ(Z* A, uint32_t sizeAB) {
    for(uint32_t i = 1; i < sizeAB; i++)
      if (A[i-1] > A[i]){
        printf("INVALID RESULT for i:%d, (A[i-1]=%d > A[i]=%d)\n", i, A[i-1], A[i]);
        return false;
      }
    return true;
}

void randomInitNat(uint32_t* data, const uint32_t size, const uint32_t H) {
    for (int i = 0; i < size; ++i) {
        unsigned long int r = rand();
        data[i] = r % H;
    }
}

double sortRedByKeyCUB( uint64_t* data_keys_in
                      , uint64_t* data_keys_out
                      , const uint64_t N
) {
    int beg_bit = 0;
    int end_bit = 64;

    void * tmp_sort_mem = NULL;
    size_t tmp_sort_len = 0;

    { // sort prelude
        hipcub::DeviceRadixSort::SortKeys( tmp_sort_mem, tmp_sort_len
                                      , data_keys_in, data_keys_out
                                      , N,   beg_bit,  end_bit
                                      );
        hipMalloc(&tmp_sort_mem, tmp_sort_len);
    }
    cudaCheckError();

    { // one dry run
        hipcub::DeviceRadixSort::SortKeys( tmp_sort_mem, tmp_sort_len
                                      , data_keys_in, data_keys_out
                                      , N,   beg_bit,  end_bit
                                      );
        hipDeviceSynchronize();
    }
    cudaCheckError();

    // timing
    double elapsed;
    struct timeval t_start, t_end, t_diff;
    gettimeofday(&t_start, NULL);

    for(int k=0; k<GPU_RUNS; k++) {
        hipcub::DeviceRadixSort::SortKeys( tmp_sort_mem, tmp_sort_len
                                      , data_keys_in, data_keys_out
                                      , N,   beg_bit,  end_bit
                                      );
    }
    hipDeviceSynchronize();
    cudaCheckError();

    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / ((double)GPU_RUNS);

    hipFree(tmp_sort_mem);

    return elapsed;
}


int main (int argc, char * argv[]) {
    for (int i=10; i<=20; i++)
    {
    int n_el = pow((double)2, (double)i);
    //Allocate and Initialize Host data with random values
    uint64_t* h_keys  = (uint64_t*) malloc(n_el*sizeof(uint64_t));
    uint64_t* h_keys_res  = (uint64_t*) malloc(n_el*sizeof(uint64_t));
    //randomInitNat(h_keys, N, N/10);

    FILE *fptr;

    fptr = fopen("../../../../IBR-Bitonic-sort/datasets/ints/random_uniform.txt", "r");
    for (int j=0; j< n_el; j++)
    {
            fscanf(fptr, "%ld", &h_keys[j]);
    };
    fclose(fptr);

    //Allocate and Initialize Device data
    uint64_t* d_keys_in;
    uint64_t* d_keys_out;
    cudaSucceeded(hipMalloc((void**) &d_keys_in,  n_el * sizeof(uint64_t)));
    cudaSucceeded(hipMemcpy(d_keys_in, h_keys, n_el * sizeof(uint64_t), hipMemcpyHostToDevice));
    cudaSucceeded(hipMalloc((void**) &d_keys_out, n_el * sizeof(uint64_t)));

    double elapsed = sortRedByKeyCUB( d_keys_in, d_keys_out, n_el );

    hipMemcpy(h_keys_res, d_keys_out, n_el*sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    cudaCheckError();

    bool success = validateZ<uint64_t>(h_keys_res, n_el);

    printf("CUB Sorting for N=%lu runs in: %.2f us, VALID: %d\n", n_el, elapsed, success);

    // Cleanup and closing
    hipFree(d_keys_in); hipFree(d_keys_out);
    free(h_keys); free(h_keys_res);
    }
    return 0;
}
