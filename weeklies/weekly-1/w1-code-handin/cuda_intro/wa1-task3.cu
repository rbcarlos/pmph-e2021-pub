#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <time.h>

unsigned int N = 1e6;
unsigned int mem_size = N * sizeof(float);
unsigned int block_size = 256;
unsigned int num_blocks = ((N + ( block_size + 1)) / block_size);

int interval_subtract( struct timeval* result, struct timeval* t2, struct timeval* t1) {
    unsigned int resolution = 1000000;
    long int diff = (t2->tv_usec + resolution * t2->tv_sec) - (t1->tv_usec + resolution * t1-> tv_sec);
    result->tv_sec = diff/resolution;
    result->tv_usec = diff%resolution;
    return (diff<0); 
}

int validate(float* h_out, float* cpu_out, float eps, int N) {
    for(int i=0; i<N; i++) {
        if(fabs(h_out[i] - cpu_out[i]) > eps) {
            printf("%.2f from gpu and %.2f from cpu", h_out[i], cpu_out[i]);
            return 1;
        }
    }
    return 0;
}

void simpleCPU(float* h_in, float *h_out, int N) {
    for(int i =0; i<N; i++){
        h_out[i] = (h_in[i] / (h_in[i] - 2.3)) * (h_in[i] / (h_in[i] - 2.3)) * (h_in[i] / (h_in[i] - 2.3));
    }
} 

__global__ void simpleKernel(float* d_in, float *d_out, int N) {
    const unsigned int lid = threadIdx.x;
    const unsigned int gid = blockIdx.x*blockDim.x + lid;
    if (gid < N) {
        d_out[gid] = (d_in[gid] / (d_in[gid] - 2.3)) * (d_in[gid] / (d_in[gid] - 2.3)) * (d_in[gid] / (d_in[gid] - 2.3)); //(x/(x-2.3))^3
    }
}

int main(int argc, char** argv) {
    unsigned long int elapsed_gpu, elapsed_cpu; struct timeval t_start, t_end, t_diff;
    // allocate host memory
    float* h_in  = (float*) malloc(mem_size);
    float* h_out = (float*) malloc(mem_size);

    float* cpu_in = (float*) malloc(mem_size);
    float* cpu_out = (float*) malloc(mem_size);

    // initialize the memory
    for(unsigned int i=0; i<N; ++i) {
        h_in[i] = (float)(i+1);
        cpu_in[i] = (float)(i+1);
    }

    // allocate device memory
    float* d_in;
    float* d_out;
    hipMalloc((void**)&d_in,  mem_size);
    hipMalloc((void**)&d_out, mem_size);

    // copy host memory to device
    hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

    gettimeofday(&t_start, NULL);

    for(int i=0; i<10; i++){
        // execute the kernel
        simpleKernel<<< num_blocks, block_size>>>(d_in, d_out, N);
    } hipDeviceSynchronize();
    
    gettimeofday(&t_end, NULL);
    interval_subtract(&t_diff, &t_end, &t_start);

    // copy result from ddevice to host
    hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);

    elapsed_gpu = (t_diff.tv_sec * 1e6 + t_diff.tv_usec) / 10;
    printf("Average GPU run took %d microseconds (%.2fms)\n", elapsed_gpu, elapsed_gpu / 1000.0);

    gettimeofday(&t_start, NULL);
    for(int i=0; i < 10; i++){
        simpleCPU(cpu_in, cpu_out, N);
    }

    gettimeofday(&t_end, NULL);
    interval_subtract(&t_diff, &t_end, &t_start);
    elapsed_cpu = (t_diff.tv_sec * 1e6 + t_diff.tv_usec) / 10;
    printf("Average CPU run took %d microseconds (%.2fms)\n", elapsed_cpu, (elapsed_cpu / 1000.0));
    //printf("Achieved speedup of %.2f\n", elapsed_cpu / elapsed_gpu);

    float eps = 0.0001;
    if (validate(h_out, cpu_out, eps, N) == 1) {
        printf("INVALID\n");
    }else {
        printf("VALID\n");
    }

    // clean-up memory
    free(h_in);       free(h_out);
    free(cpu_in);     free(cpu_out);
    hipFree(d_in);   hipFree(d_out);
}
